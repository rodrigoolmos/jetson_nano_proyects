
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N_ITEMS 1024*1024*128


// Kernel para la suma de vectores en CUDA
__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

void vector_add(float *a, float *b, float *c, int n){

    hipEvent_t start1, stop1;
    float milliseconds = 0;

    hipEventCreate(&start1);
    hipEventCreate(&stop1);

    hipEventRecord(start1, 0);    
    for (int i = 0; i < n; i++){
        c[i] = a[i] + b[i];
    }
    hipEventRecord(stop1, 0);
 
    hipEventElapsedTime(&milliseconds, start1, stop1);
    printf("Tiempo de ejecución CLP: %.4f ms\n", milliseconds);

}

int main() {

    hipEvent_t start1, stop1;
    hipEvent_t start2, stop2;

    hipEventCreate(&start1);
    hipEventCreate(&start2);
    hipEventCreate(&stop1);
    hipEventCreate(&stop2);
    hipDeviceProp_t deviceProp;

    int blockSize;
    int gridSize;

    float milliseconds1 = 0;
    float milliseconds2 = 0;


    float *d_a, *d_b, *d_c;

    // Alojar memoria en el host
    float *h_a = (float*)malloc(sizeof(float) * N_ITEMS);
    float *h_b = (float*)malloc(sizeof(float) * N_ITEMS);
    float *h_c = (float*)malloc(sizeof(float) * N_ITEMS);

    hipGetDeviceProperties(&deviceProp, 0);

    printf("Device %s\n", deviceProp.name);
    printf("Max threads per SM: %d\n", deviceProp.maxThreadsPerMultiProcessor);

    // Inicializar los vectores h_a y h_b
    for (int i = 0; i < N_ITEMS; i++) {
        h_a[i] = i;
        h_b[i] = i;
    }

    // Alojar memoria en el dispositivo
    hipMalloc((void**)&d_a, sizeof(float) * N_ITEMS);
    hipMalloc((void**)&d_b, sizeof(float) * N_ITEMS);
    hipMalloc((void**)&d_c, sizeof(float) * N_ITEMS);

    // Configurar los parámetros del kernel
    blockSize = 1024;
    gridSize = (N_ITEMS + blockSize - 1) / blockSize;

    // Transferir datos desde el host al dispositivo
    hipEventRecord(start1, 0);    
    hipMemcpy(d_a, h_a, sizeof(float) * N_ITEMS, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(float) * N_ITEMS, hipMemcpyHostToDevice);
    // Llamar al kernel
    hipEventRecord(start2, 0);    
    vectorAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, N_ITEMS);
    hipEventRecord(stop2, 0);
    // Transferir resultado desde el dispositivo al host
    hipMemcpy(h_c, d_c, sizeof(float) * N_ITEMS, hipMemcpyDeviceToHost);
    hipEventRecord(stop1, 0);

    printf("Resultado de la suma:\n");
    for (int i = 0; i < 10; i++) {
        printf("%.2f + %.2f = %.2f\n", h_a[i], h_b[i], h_c[i]);
    }

    hipEventElapsedTime(&milliseconds1, start1, stop1);
    hipEventElapsedTime(&milliseconds2, start2, stop2);

    printf("Tiempo de ejecución del kernel: %.4f ms tiempo total  %.4f ms\n", milliseconds2, milliseconds1);

    vector_add(h_a, h_b, h_c, N_ITEMS);

    // Liberar memoria
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
