#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N (1024 * 2)
// Coalescing Factor
#define COARSE_FACTOR 8
// Tiles of A
#define tiles_A_rows 64
#define tiles_A_cols 8

// Tiles of B
#define tiles_B_cols 64

__global__ void matmul_gpu(float *d_A, float *d_B, float *d_C)
{
    // Details regarding this thread
    const int by = blockIdx.y;
    const int bx = blockIdx.x;

    const int tx = threadIdx.x;

    // 1D -> 2D while loading A
    const int A_view_ty = tx / tiles_A_cols;
    const int A_view_tx = tx % tiles_A_cols;

    // 1D -> 2D while loading B
    const int B_view_ty = tx / tiles_B_cols;
    const int B_view_tx = tx % tiles_B_cols;

    // Working on C[row,col]
    const int row = tiles_A_rows * by + COARSE_FACTOR * (tx / tiles_B_cols);
    const int col = tiles_B_cols * bx + (tx % tiles_B_cols);

    // Allocating shared memory
    __shared__ float sh_A[tiles_A_rows][tiles_A_cols];
    __shared__ float sh_B[tiles_A_cols][tiles_B_cols];

    // Phases
    const int phases = ceil((float)N / tiles_A_cols);

    // Parallel mat mul
    float value[COARSE_FACTOR] = {0.0f};
    for (int phase = 0; phase < phases; phase++)
    {
        // Load Tiles into shared memory
        if ((by * tiles_A_rows + A_view_ty < N) && ((phase * tiles_A_cols + A_view_tx) < N))
            sh_A[A_view_ty][A_view_tx] = d_A[(by * tiles_A_rows + A_view_ty) * N + (phase * tiles_A_cols + A_view_tx)];
        else
            sh_A[A_view_ty][A_view_tx] = 0.0f;

        if (((phase * tiles_A_cols + B_view_ty) < N) && (bx * tiles_B_cols + B_view_tx < N))
            sh_B[B_view_ty][B_view_tx] = d_B[(phase * tiles_A_cols + B_view_ty) * N + (bx * tiles_B_cols + B_view_tx)];
        else
            sh_B[B_view_ty][B_view_tx] = 0.0f;
        __syncthreads();

        for (int k = 0; k < tiles_A_cols; k++)
        {
            float B_val_register = sh_B[k][B_view_tx];
            // Dot product
            for (int c = 0; c < COARSE_FACTOR; c++)
                value[c] += sh_A[B_view_ty * COARSE_FACTOR + c][k] * B_val_register;
        }
        __syncthreads();
    }

    // Assigning calculated value
    for (int c = 0; c < COARSE_FACTOR; ++c)
    {
        if ((row + c < N) && (col < N))
            d_C[(row + c) * N + (col)] = 1 * value[c];
    }
}

void printMatrix(const float *mat, int rows, int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%.2f ", mat[i * cols + j]);
        }
        printf("\n");
    }
}

// Multiplicación de matrices en CPU (implementación sencilla)
void matmul_cpu(float *h_A, float *h_B, float *h_C_cpu)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            float sum = 0.0f;
            for (int k = 0; k < N; k++)
            {
                sum += h_A[i * N + k] * h_B[k * N + j];
            }
            h_C_cpu[i * N + j] = sum;
        }
    }
}

int main()
{
    struct timeval start, stop;
    double elapsed;
    int error = 0;

    size_t bytes = N * N * sizeof(float);

    // Alojar memoria en la CPU
    float *h_A = (float *)malloc(bytes);
    float *h_B = (float *)malloc(bytes);
    float *h_C = (float *)malloc(bytes);
    float *h_C_cpu = (float *)malloc(bytes);

    // Inicializar matrices en la CPU
    for (int i = 0; i < N * N; i++)
    {
        h_A[i] = (rand() % 20 - 10) / 10.0f;
        h_B[i] = (rand() % 20 - 10) / 10.0f;
        h_C[i] = 0.0f;
        h_C_cpu[i] = 0.0f;
    }

    // Alojar memoria en la GPU
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, bytes);
    hipMalloc((void **)&d_B, bytes);
    hipMalloc((void **)&d_C, bytes);

    
    // Definir el tamaño del bloque y la cuadrícula en 2D
    dim3 gridSize(ceil(N / (float)(tiles_B_cols)), ceil(N / (float)(tiles_A_rows)));
    dim3 blockSize(tiles_A_rows * tiles_B_cols / COARSE_FACTOR);
    
    gettimeofday(&start, NULL);
    // Copiar datos desde la CPU a la GPU
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
    // Ejecutar kernel en GPU
    matmul_gpu<<<gridSize, blockSize>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();
    // Copiar el resultado de vuelta a la CPU
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);
    gettimeofday(&stop, NULL);

    elapsed = (stop.tv_sec - start.tv_sec) * 1000.0 +
              (stop.tv_usec - start.tv_usec) / 1000.0;
    printf("Tiempo de ejecución en la GPU: %.4f ms, tamaño %dx%d\n", elapsed, N, N);


    // Calcular el resultado en CPU (solo para verificar)
    gettimeofday(&start, NULL);
    matmul_cpu(h_A, h_B, h_C_cpu);
    gettimeofday(&stop, NULL);
    elapsed = (stop.tv_sec - start.tv_sec) * 1000.0 +
              (stop.tv_usec - start.tv_usec) / 1000.0;
    printf("Tiempo de ejecución en la CPU: %.4f ms, tamaño %dx%d\n", elapsed, N, N);

    // Verificación de resultados
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            if (fabs(h_C[i * N + j] - h_C_cpu[i * N + j]) > 0.1f)
            {
                error = 1;
                printf("Desajuste en [%d][%d]: GPU = %f, CPU = %f\n", i, j,
                       h_C[i * N + j], h_C_cpu[i * N + j]);
            }
        }
    }
    printf("Multiplicación de matrices %s\n", error ? "incorrecta" : "correcta");

    // Liberar memoria en GPU y CPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_cpu);

    return 0;
}
