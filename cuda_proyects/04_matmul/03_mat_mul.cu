#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N (1024*2)         // Tamaño de la matriz (N x N). Debe ser múltiplo del tamaño del bloque.
#define TILE_SIZE 32       // Tamaño del tile. Debe ser consistente con el tamaño del bloque.

__global__ void matmul_gpu(float *d_A, float *d_B, float *d_C) {
    // Declaración de memoria compartida para los tiles
    __shared__ float tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_B[TILE_SIZE][TILE_SIZE];

    // Índices globales para el elemento de C que calculará este hilo
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;

    // Iterar sobre todos los tiles necesarios (la cantidad de tiles en la dimensión k)
    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Índices para acceder al tile actual de A y B
        int tiledCol = t * TILE_SIZE + threadIdx.x; // para A
        int tiledRow = t * TILE_SIZE + threadIdx.y; // para B

        // Cargar el tile de A de forma coalescente
        if (row < N && tiledCol < N)
            tile_A[threadIdx.y][threadIdx.x] = d_A[row * N + tiledCol];
        else
            tile_A[threadIdx.y][threadIdx.x] = 0.0f;

        // Cargar el tile de B de forma coalescente.
        // Recordar: para la multiplicación, se requiere que B se acceda como B[k][col]
        if (col < N && tiledRow < N)
            tile_B[threadIdx.y][threadIdx.x] = d_B[tiledRow * N + col];
        else
            tile_B[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        // Cada hilo acumula la suma del producto de la fila del tile de A y la columna del tile de B
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += tile_A[threadIdx.y][k] * tile_B[k][threadIdx.x];
        }
        __syncthreads();
    }

    // Escribir el resultado en la matriz C en memoria global
    if (row < N && col < N)
        d_C[row * N + col] = sum;
}

void printMatrix(const float* mat, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%.2f ", mat[i * cols + j]);
        }
        printf("\n");
    }
}

// Multiplicación de matrices en CPU (implementación sencilla)
void matmul_cpu(float *h_A, float *h_B, float *h_C_cpu) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            float sum = 0.0f;
            for (int k = 0; k < N; k++) {
                sum += h_A[i * N + k] * h_B[k * N + j];
            }
            h_C_cpu[i * N + j] = sum;
        }
    }
}

int main() {
    struct timeval start, stop;
    double elapsed;
    int error = 0;

    size_t bytes = N * N * sizeof(float);

    // Alojar memoria en la CPU
    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);
    float *h_C_cpu = (float*)malloc(bytes);

    // Inicializar matrices en la CPU
    for (int i = 0; i < N * N; i++) {
        h_A[i] = (rand() % 20 - 10) / 10.0f;
        h_B[i] = (rand() % 20 - 10) / 10.0f;
        h_C[i] = 0.0f;
        h_C_cpu[i] = 0.0f;
    }

    // Alojar memoria en la GPU
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, bytes);
    hipMalloc((void**)&d_B, bytes);
    hipMalloc((void**)&d_C, bytes);
    
    // Definir el tamaño del bloque y la cuadrícula en 2D
    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);
    
    gettimeofday(&start, NULL);
    // Copiar datos desde la CPU a la GPU
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
    // Ejecutar kernel en GPU
    matmul_gpu<<<gridSize, blockSize>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();
    
    // Copiar el resultado de vuelta a la CPU
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);
    
    gettimeofday(&stop, NULL);

    elapsed = (stop.tv_sec - start.tv_sec) * 1000.0 +
              (stop.tv_usec - start.tv_usec) / 1000.0;
    printf("Tiempo de ejecución en la GPU: %.4f ms, tamaño %dx%d\n", elapsed, N, N);

    // Calcular el resultado en CPU (solo para verificar)
    gettimeofday(&start, NULL);
    matmul_cpu(h_A, h_B, h_C_cpu);
    gettimeofday(&stop, NULL);
    elapsed = (stop.tv_sec - start.tv_sec) * 1000.0 +
              (stop.tv_usec - start.tv_usec) / 1000.0;
    printf("Tiempo de ejecución en la CPU: %.4f ms, tamaño %dx%d\n", elapsed, N, N);

    // Verificación de resultados
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (fabs(h_C[i * N + j] - h_C_cpu[i * N + j]) > 0.1f) {
                error = 1;
                printf("Desajuste en [%d][%d]: GPU = %f, CPU = %f\n", i, j,
                       h_C[i * N + j], h_C_cpu[i * N + j]);
            }
        }
    }
    printf("Multiplicación de matrices %s\n", error ? "incorrecta" : "correcta");

    // Liberar memoria en GPU y CPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_cpu);

    return 0;
}
