#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N (1024 * 2)
// Coalescing Factor
#define COARSE_FACTOR_X 8
#define COARSE_FACTOR_Y 8

// Tiles of A
#define tiles_A_rows 64
#define tiles_A_cols 8

// Tiles of B
#define tiles_B_cols 64

__global__ void matmul_gpu(float *d_A_ptr, float *d_B_ptr, float *d_C_ptr)
{
    // Número de hilos por bloque
    const int n_threads_per_block = tiles_A_rows * tiles_B_cols / (COARSE_FACTOR_X * COARSE_FACTOR_Y);
    static_assert(n_threads_per_block % tiles_A_cols == 0);
    static_assert(n_threads_per_block % tiles_B_cols == 0);
    static_assert(tiles_A_cols % 4 == 0);
    static_assert(tiles_B_cols % 4 == 0);

    // Datos del bloque e hilo
    const int by = blockIdx.y;
    const int bx = blockIdx.x;
    const int tx = threadIdx.x;

    // Conversión de 1D a 2D para cargar A
    const int A_view_ty = tx / (tiles_A_cols / 4);
    const int A_view_tx = tx % (tiles_A_cols / 4);
    const int stride_A = n_threads_per_block / (tiles_A_cols / 4);

    // Conversión de 1D a 2D para cargar B
    const int B_view_ty = tx / (tiles_B_cols / 4);
    const int B_view_tx = tx % (tiles_B_cols / 4);
    const int stride_B = n_threads_per_block / (tiles_B_cols / 4);

    // Elemento de C que calcula este hilo
    const int row = COARSE_FACTOR_Y * (tx / (tiles_B_cols / COARSE_FACTOR_X));
    const int col = COARSE_FACTOR_X * (tx % (tiles_B_cols / COARSE_FACTOR_X));

    // Memoria compartida para tiles
    __shared__ float sh_A[tiles_A_cols][tiles_A_rows];
    __shared__ float sh_B[tiles_A_cols][tiles_B_cols];

    // Acumuladores en registros
    float value[COARSE_FACTOR_Y][COARSE_FACTOR_X] = {0.0f};
    float register_A[COARSE_FACTOR_Y] = {0.0f};
    float register_B[COARSE_FACTOR_X] = {0.0f};

    // Número de fases (tiles) a recorrer
    const int phases = ceil((float)N / tiles_A_cols);

    for (int phase = 0; phase < phases; phase++)
    {
        // Cargar tiles de A en memoria compartida
        for (int load_offset = 0; load_offset < tiles_A_rows; load_offset += stride_A)
        {
            if ((by * tiles_A_rows + load_offset + A_view_ty < N) &&
                ((phase * tiles_A_cols + A_view_tx * 4) < N))
            {
                float4 A_tmp = reinterpret_cast<float4 *>(
                    &d_A_ptr[(by * tiles_A_rows + load_offset + A_view_ty) * N +
                             (phase * tiles_A_cols + A_view_tx * 4)])[0];
                sh_A[A_view_tx * 4 + 0][load_offset + A_view_ty] = A_tmp.x;
                sh_A[A_view_tx * 4 + 1][load_offset + A_view_ty] = A_tmp.y;
                sh_A[A_view_tx * 4 + 2][load_offset + A_view_ty] = A_tmp.z;
                sh_A[A_view_tx * 4 + 3][load_offset + A_view_ty] = A_tmp.w;
            }
            else
            {
                sh_A[A_view_tx * 4 + 0][load_offset + A_view_ty] = 0.0f;
                sh_A[A_view_tx * 4 + 1][load_offset + A_view_ty] = 0.0f;
                sh_A[A_view_tx * 4 + 2][load_offset + A_view_ty] = 0.0f;
                sh_A[A_view_tx * 4 + 3][load_offset + A_view_ty] = 0.0f;
            }
        }

        // Cargar tiles de B en memoria compartida
        for (int load_offset = 0; load_offset < tiles_A_cols; load_offset += stride_B)
        {
            if (((phase * tiles_A_cols + B_view_ty + load_offset) < N) &&
                ((bx * tiles_B_cols + B_view_tx * 4) < N))
            {
                float4 B_tmp = reinterpret_cast<float4 *>(
                    &d_B_ptr[(phase * tiles_A_cols + B_view_ty + load_offset) * N +
                             (bx * tiles_B_cols + B_view_tx * 4)])[0];
                sh_B[B_view_ty + load_offset][B_view_tx * 4 + 0] = B_tmp.x;
                sh_B[B_view_ty + load_offset][B_view_tx * 4 + 1] = B_tmp.y;
                sh_B[B_view_ty + load_offset][B_view_tx * 4 + 2] = B_tmp.z;
                sh_B[B_view_ty + load_offset][B_view_tx * 4 + 3] = B_tmp.w;
            }
            else
            {
                sh_B[B_view_ty + load_offset][B_view_tx * 4 + 0] = 0.0f;
                sh_B[B_view_ty + load_offset][B_view_tx * 4 + 1] = 0.0f;
                sh_B[B_view_ty + load_offset][B_view_tx * 4 + 2] = 0.0f;
                sh_B[B_view_ty + load_offset][B_view_tx * 4 + 3] = 0.0f;
            }
        }
        __syncthreads();

        // Calcular el producto parcial
        for (int k = 0; k < tiles_A_cols; ++k)
        {
            // Cargar datos en registros
            for (int i = 0; i < COARSE_FACTOR_Y; ++i)
                register_A[i] = sh_A[k][row + i];
            for (int i = 0; i < COARSE_FACTOR_X; ++i)
                register_B[i] = sh_B[k][col + i];

            for (int cy = 0; cy < COARSE_FACTOR_Y; ++cy)
            {
                for (int cx = 0; cx < COARSE_FACTOR_X; ++cx)
                    value[cy][cx] += register_A[cy] * register_B[cx];
            }
        }
        __syncthreads();
    }

    // Escribir el resultado en memoria global
    for (int cy = 0; cy < COARSE_FACTOR_Y; ++cy)
    {
        for (int cx = 0; cx < COARSE_FACTOR_X; cx++)
        {
            if ((by * tiles_A_rows + row + cy < N) && (bx * tiles_B_cols + col + cx < N))
                d_C_ptr[(by * tiles_A_rows + row + cy) * N +
                        (bx * tiles_B_cols + col + cx)] = value[cy][cx];
        }
    }
}

void printMatrix(const float *mat, int rows, int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%.2f ", mat[i * cols + j]);
        }
        printf("\n");
    }
}

// Multiplicación de matrices en CPU (implementación sencilla)
void matmul_cpu(float *h_A, float *h_B, float *h_C_cpu)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            float sum = 0.0f;
            for (int k = 0; k < N; k++)
            {
                sum += h_A[i * N + k] * h_B[k * N + j];
            }
            h_C_cpu[i * N + j] = sum;
        }
    }
}

int main()
{
    struct timeval start, stop;
    double elapsed;
    int error = 0;
    size_t bytes = N * N * sizeof(float);

    // Alojar memoria en el host usando memoria pinned mapeada (zero copy)
    float *h_A, *h_B, *h_C;
    hipHostAlloc((void **)&h_A, bytes, hipHostMallocMapped);
    hipHostAlloc((void **)&h_B, bytes, hipHostMallocMapped);
    hipHostAlloc((void **)&h_C, bytes, hipHostMallocMapped);
    // Para la verificación se usa memoria normal en CPU
    float *h_C_cpu = (float *)malloc(bytes);

    // Inicializar las matrices en el host
    for (int i = 0; i < N * N; i++)
    {
        h_A[i] = (rand() % 20 - 10) / 10.0f;
        h_B[i] = (rand() % 20 - 10) / 10.0f;
        h_C[i] = 0.0f;
        h_C_cpu[i] = 0.0f;
    }

    // Obtener los punteros de dispositivo para la memoria mapeada
    float *d_A, *d_B, *d_C;
    hipHostGetDevicePointer((void **)&d_A, h_A, 0);
    hipHostGetDevicePointer((void **)&d_B, h_B, 0);
    hipHostGetDevicePointer((void **)&d_C, h_C, 0);

    // Definir el tamaño del bloque y de la cuadrícula en 2D
    dim3 gridSize(ceil(N / (float)(tiles_B_cols)), ceil(N / (float)(tiles_A_rows)));
    dim3 blockSize(tiles_A_rows * tiles_B_cols / (COARSE_FACTOR_X * COARSE_FACTOR_Y));

    gettimeofday(&start, NULL);
    // Ejecutar el kernel en la GPU (al usar zero copy no se requieren cudaMemcpy)
    matmul_gpu<<<gridSize, blockSize>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();
    gettimeofday(&stop, NULL);
    elapsed = (stop.tv_sec - start.tv_sec) * 1000.0 +
              (stop.tv_usec - start.tv_usec) / 1000.0;
    printf("Tiempo de ejecución en la GPU: %.4f ms, tamaño %dx%d\n", elapsed, N, N);

    // h_C ya contiene el resultado calculado por el GPU

    // Calcular el resultado en CPU para verificación
    gettimeofday(&start, NULL);
    matmul_cpu(h_A, h_B, h_C_cpu);
    gettimeofday(&stop, NULL);
    elapsed = (stop.tv_sec - start.tv_sec) * 1000.0 +
              (stop.tv_usec - start.tv_usec) / 1000.0;
    printf("Tiempo de ejecución en la CPU: %.4f ms, tamaño %dx%d\n", elapsed, N, N);

    // Verificar resultados
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            if (fabs(h_C[i * N + j] - h_C_cpu[i * N + j]) > 0.1f)
            {
                error = 1;
                printf("Desajuste en [%d][%d]: GPU = %f, CPU = %f\n", i, j,
                       h_C[i * N + j], h_C_cpu[i * N + j]);
            }
        }
    }
    printf("Multiplicación de matrices %s\n", error ? "incorrecta" : "correcta");

    // Liberar la memoria
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    free(h_C_cpu);

    return 0;
}
